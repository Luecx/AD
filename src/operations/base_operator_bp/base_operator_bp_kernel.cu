#include "hip/hip_runtime.h"
/**
    CudAD is a CUDA neural network trainer, specific for chess engines.
    Copyright (C) 2022 Finn Eggers

    This program is free software: you can redistribute it and/or modify
    it under the terms of the GNU General Public License as published by
    the Free Software Foundation, either version 3 of the License, or
    (at your option) any later version.

    This program is distributed in the hope that it will be useful,
    but WITHOUT ANY WARRANTY; without even the implied warranty of
    MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
    GNU General Public License for more details.

    You should have received a copy of the GNU General Public License
    along with this program.  If not, see <https://www.gnu.org/licenses/>.
 */

#include "base_operator_bp.h"

// clang-format off
template<BaseOperation operation>
__global__ void base_operator_bp_kernel(
    const float* __restrict__ A,
          float* __restrict__ A_grd,
    const float* __restrict__ B,
          float* __restrict__ B_grd,
    const float* __restrict__ C_grd,
    unsigned int              size){
    // clang-format on

    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx >= size)
        return;

    if constexpr (operation == BASE_OPERATOR_OP_ADD) {
        A_grd[idx] += C_grd[idx];
        B_grd[idx] += C_grd[idx];
    } else if constexpr (operation == BASE_OPERATOR_OP_SUB) {
        A_grd[idx] += C_grd[idx];
        B_grd[idx] -= C_grd[idx];
    } else if constexpr (operation == BASE_OPERATOR_OP_DIV) {
        A_grd[idx] += C_grd[idx] / B[idx];
        B_grd[idx] -= C_grd[idx] * A[idx] / (B[idx] * B[idx]);
    } else if constexpr (operation == BASE_OPERATOR_OP_MUL) {
        A_grd[idx] += C_grd[idx] * B[idx];
        B_grd[idx] += C_grd[idx] * A[idx];
    }
}

template void __global__
    base_operator_bp_kernel<BASE_OPERATOR_OP_ADD>(const float* __restrict__ A,
                                                  float* __restrict__ A_grd,
                                                  const float* __restrict__ B,
                                                  float* __restrict__ B_grd,
                                                  const float* __restrict__ C_grd,
                                                  unsigned int size);
template void __global__
    base_operator_bp_kernel<BASE_OPERATOR_OP_SUB>(const float* __restrict__ A,
                                                  float* __restrict__ A_grd,
                                                  const float* __restrict__ B,
                                                  float* __restrict__ B_grd,
                                                  const float* __restrict__ C_grd,
                                                  unsigned int size);
template void __global__
    base_operator_bp_kernel<BASE_OPERATOR_OP_DIV>(const float* __restrict__ A,
                                                  float* __restrict__ A_grd,
                                                  const float* __restrict__ B,
                                                  float* __restrict__ B_grd,
                                                  const float* __restrict__ C_grd,
                                                  unsigned int size);
template void __global__
    base_operator_bp_kernel<BASE_OPERATOR_OP_MUL>(const float* __restrict__ A,
                                                  float* __restrict__ A_grd,
                                                  const float* __restrict__ B,
                                                  float* __restrict__ B_grd,
                                                  const float* __restrict__ C_grd,
                                                  unsigned int size);