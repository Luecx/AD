
#include <hip/hip_runtime.h>
/**
    CudAD is a CUDA neural network trainer, specific for chess engines.
    Copyright (C) 2022 Finn Eggers

    This program is free software: you can redistribute it and/or modify
    it under the terms of the GNU General Public License as published by
    the Free Software Foundation, either version 3 of the License, or
    (at your option) any later version.

    This program is distributed in the hope that it will be useful,
    but WITHOUT ANY WARRANTY; without even the implied warranty of
    MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
    GNU General Public License for more details.

    You should have received a copy of the GNU General Public License
    along with this program.  If not, see <https://www.gnu.org/licenses/>.
 */

// https://github.com/LiyuanLucasLiu/RAdam/blob/master/radam/radam.py#L96

// clang-format off
__global__ void radam_kernel(
          float* __restrict__ values,
          float* __restrict__ gradients,
          float* __restrict__ exp_avg,
          float* __restrict__ exp_avg_sq,
          int   size,
          int   step,
          float lr,
          float beta1,
          float beta2,
          float eps,
          int   N_sma_threshold) {

    // clang-format on
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= size)
        return;

    exp_avg_sq[idx] = beta2 * exp_avg_sq[idx] + (1.0 - beta2) * gradients[idx] * gradients[idx];
    exp_avg[idx]    = beta1 * exp_avg[idx] + (1.0 - beta1) * gradients[idx];

    // we increment step in the struct, no need to do it here

    float beta2_t   = powf(beta2, step);
    float N_sma_max = 2.0 / (1.0 - beta2) - 1.0;
    float N_sma     = N_sma_max - 2 * step * beta2_t / (1.0 - beta2_t);

    if (N_sma >= N_sma_threshold) {
        float step_size = lr
                          * sqrtf((1.0 - beta2_t) * (N_sma - 4.0) / (N_sma_max - 4.0) * (N_sma - 2.0)
                                  / N_sma * N_sma_max / (N_sma_max - 2.0))
                          / (1.0 - powf(beta1, step));

        float denom = sqrtf(exp_avg_sq[idx]) + eps;
        float delta = step_size * exp_avg[idx] / denom;

        values[idx] -= delta;
    } else {
        float step_size = lr * (1.0 - powf(beta1, step));
        float delta     = step_size * exp_avg[idx];

        values[idx] -= delta;
    }

    gradients[idx] = 0;
}