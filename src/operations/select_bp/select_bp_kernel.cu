#include "hip/hip_runtime.h"
/**
    CudAD is a CUDA neural network trainer, specific for chess engines.
    Copyright (C) 2022 Finn Eggers

    This program is free software: you can redistribute it and/or modify
    it under the terms of the GNU General Public License as published by
    the Free Software Foundation, either version 3 of the License, or
    (at your option) any later version.

    This program is distributed in the hope that it will be useful,
    but WITHOUT ANY WARRANTY; without even the implied warranty of
    MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
    GNU General Public License for more details.

    You should have received a copy of the GNU General Public License
    along with this program.  If not, see <https://www.gnu.org/licenses/>.
 */

#include "select_bp.h"

// clang-format off
__global__ void select_bp_kernel(
          float* __restrict__ A_grd,
    const int  * __restrict__ index,
    const float* __restrict__ B_grd,
    unsigned int m,
    unsigned int n){
    // clang-format on

    int idy = blockIdx.y * blockDim.y + threadIdx.y;

    if (idy >= m)
        return;

    A_grd[INDEX_2D(n,idy, index[idy])] += B_grd[idy];
}
