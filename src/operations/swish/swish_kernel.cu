#include "hip/hip_runtime.h"
/**
    CudAD is a CUDA neural network trainer, specific for chess engines.
    Copyright (C) 2022 Finn Eggers

    This program is free software: you can redistribute it and/or modify
    it under the terms of the GNU General Public License as published by
    the Free Software Foundation, either version 3 of the License, or
    (at your option) any later version.

    This program is distributed in the hope that it will be useful,
    but WITHOUT ANY WARRANTY; without even the implied warranty of
    MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
    GNU General Public License for more details.

    You should have received a copy of the GNU General Public License
    along with this program.  If not, see <https://www.gnu.org/licenses/>.
 */

#include "swish.h"

// clang-format off
__global__ void swish_kernel(
    const float* __restrict__ A,
          float* __restrict__ B,
    unsigned int size,
    float scalar){
    // clang-format on

    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx >= size)
        return;

    B[idx] = A[idx] *  1.0f / (1.0f + expf(-A[idx] * scalar));
}
