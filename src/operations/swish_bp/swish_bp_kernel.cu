#include "hip/hip_runtime.h"
/**
    CudAD is a CUDA neural network trainer, specific for chess engines.
    Copyright (C) 2022 Finn Eggers

    This program is free software: you can redistribute it and/or modify
    it under the terms of the GNU General Public License as published by
    the Free Software Foundation, either version 3 of the License, or
    (at your option) any later version.

    This program is distributed in the hope that it will be useful,
    but WITHOUT ANY WARRANTY; without even the implied warranty of
    MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
    GNU General Public License for more details.

    You should have received a copy of the GNU General Public License
    along with this program.  If not, see <https://www.gnu.org/licenses/>.
 */

#include "swish_bp.h"

// clang-format off
__global__ void swish_bp_kernel(
    const float* __restrict__ A,
          float* __restrict__ A_grd,
    const float* __restrict__ B,
    const float* __restrict__ B_grd,
    unsigned int size,
    float scalar){
    // clang-format on

    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx >= size)
        return;

    A_grd[idx] += B_grd[idx] * (
                      B[idx] / A[idx] + B[idx] * B[idx] / A[idx] * scalar * exp(-scalar * A[idx]));
}
